/*
 * CUDA Kernel for Parallel Chess Constraint Checking
 * Each thread evaluates one possible piece placement
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

// Board constants
#define MAX_BOARD_SIZE 9
#define MAX_PIECES 8

// Piece types
#define KING 0
#define QUEEN 1
#define BISHOP 2
#define KNIGHT 3
#define ROOK 4

// Compact board representation using bit vectors
typedef struct {
    int m, n;                    // Board dimensions
    unsigned long long occupied; // Occupied slots as bit vector (64 bits max)
    char pieces[MAX_PIECES];     // Piece types to place
    char positions[MAX_PIECES];  // Current piece positions (encoded as x*n+y)
    int piece_count;             // Number of pieces
    int current_piece;           // Current piece being placed
} BoardState;

// Device function to check if two pieces attack each other
__device__ bool pieces_attack(int piece1_type, int pos1, int piece2_type, int pos2, int m, int n) {
    int x1 = pos1 / n, y1 = pos1 % n;
    int x2 = pos2 / n, y2 = pos2 % n;
    
    // Same position
    if (pos1 == pos2) return true;
    
    // Helper function to check if position is valid
    auto valid = [m, n](int x, int y) { return x >= 0 && x < m && y >= 0 && y < n; };
    
    // King attacks
    if (piece1_type == KING) {
        int dx = abs(x2 - x1);
        int dy = abs(y2 - y1);
        if (dx <= 1 && dy <= 1 && (dx + dy > 0)) return true;
    }
    
    // Queen attacks (rook + bishop)
    if (piece1_type == QUEEN) {
        // Horizontal/vertical (rook-like)
        if (x1 == x2 || y1 == y2) return true;
        // Diagonal (bishop-like)
        if (abs(x2 - x1) == abs(y2 - y1)) return true;
    }
    
    // Bishop attacks
    if (piece1_type == BISHOP) {
        if (abs(x2 - x1) == abs(y2 - y1)) return true;
    }
    
    // Knight attacks
    if (piece1_type == KNIGHT) {
        int dx = abs(x2 - x1);
        int dy = abs(y2 - y1);
        if ((dx == 2 && dy == 1) || (dx == 1 && dy == 2)) return true;
    }
    
    return false;
}

// Device function to check if a board state is valid
__device__ bool is_valid_board(BoardState* state) {
    // Check all pairs of placed pieces
    for (int i = 0; i < state->current_piece; i++) {
        for (int j = i + 1; j < state->current_piece; j++) {
            if (pieces_attack(state->pieces[i], state->positions[i], 
                            state->pieces[j], state->positions[j], 
                            state->m, state->n)) {
                return false;
            }
        }
    }
    return true;
}

// CUDA kernel for parallel board evaluation
__global__ void evaluate_boards(BoardState* input_states, bool* results, int num_states) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < num_states) {
        results[tid] = is_valid_board(&input_states[tid]);
    }
}

// CUDA kernel for massively parallel placement generation
__global__ void generate_placements(
    BoardState* base_state,     // Input: base board state
    int* available_positions,   // Available positions to try
    int num_positions,          // Number of available positions
    BoardState* output_states,  // Output: generated board states
    bool* valid_flags,          // Output: validity flags
    int piece_to_place         // Which piece we're placing
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < num_positions) {
        // Copy base state
        output_states[tid] = *base_state;
        
        // Place piece at this position
        output_states[tid].positions[piece_to_place] = available_positions[tid];
        output_states[tid].current_piece = piece_to_place + 1;
        
        // Check if placement is valid
        valid_flags[tid] = is_valid_board(&output_states[tid]);
    }
}

// CUDA kernel for symmetry elimination (canonical form computation)
__global__ void compute_canonical_forms(
    BoardState* states,
    unsigned long long* canonical_hashes,
    int num_states,
    int m, int n
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < num_states) {
        BoardState* state = &states[tid];
        unsigned long long min_hash = ULLONG_MAX;
        
        // Try all 8 symmetries and find minimum hash
        for (int sym = 0; sym < 8; sym++) {
            unsigned long long hash = 0;
            
            for (int piece_idx = 0; piece_idx < state->current_piece; piece_idx++) {
                int pos = state->positions[piece_idx];
                int x = pos / n, y = pos % n;
                int new_x, new_y;
                
                // Apply symmetry transformation
                switch (sym) {
                    case 0: new_x = x; new_y = y; break;                    // Identity
                    case 1: new_x = n-1-y; new_y = x; break;               // 90° rotation
                    case 2: new_x = m-1-x; new_y = n-1-y; break;           // 180° rotation
                    case 3: new_x = y; new_y = m-1-x; break;               // 270° rotation
                    case 4: new_x = m-1-x; new_y = y; break;               // Horizontal reflection
                    case 5: new_x = x; new_y = n-1-y; break;               // Vertical reflection
                    case 6: new_x = n-1-y; new_y = m-1-x; break;           // Diagonal reflection
                    case 7: new_x = y; new_y = x; break;                   // Anti-diagonal reflection
                }
                
                int new_pos = new_x * n + new_y;
                
                // Contribute to hash (simple polynomial hash)
                hash = hash * 31 + (state->pieces[piece_idx] * 100 + new_pos);
            }
            
            min_hash = min(min_hash, hash);
        }
        
        canonical_hashes[tid] = min_hash;
    }
}

// Host function prototypes (to be called from JCuda)
extern "C" {
    void launch_evaluate_boards(BoardState* states, bool* results, int num_states, hipStream_t stream);
    void launch_generate_placements(BoardState* base, int* positions, int num_pos, 
                                   BoardState* output, bool* valid, int piece, hipStream_t stream);
    void launch_canonical_forms(BoardState* states, unsigned long long* hashes, 
                               int num_states, int m, int n, hipStream_t stream);
}

void launch_evaluate_boards(BoardState* states, bool* results, int num_states, hipStream_t stream) {
    int block_size = 256;
    int grid_size = (num_states + block_size - 1) / block_size;
    evaluate_boards<<<grid_size, block_size, 0, stream>>>(states, results, num_states);
}

void launch_generate_placements(BoardState* base, int* positions, int num_pos, 
                               BoardState* output, bool* valid, int piece, hipStream_t stream) {
    int block_size = 256;
    int grid_size = (num_pos + block_size - 1) / block_size;
    generate_placements<<<grid_size, block_size, 0, stream>>>(base, positions, num_pos, output, valid, piece);
}

void launch_canonical_forms(BoardState* states, unsigned long long* hashes, 
                           int num_states, int m, int n, hipStream_t stream) {
    int block_size = 256;
    int grid_size = (num_states + block_size - 1) / block_size;
    compute_canonical_forms<<<grid_size, block_size, 0, stream>>>(states, hashes, num_states, m, n);
}